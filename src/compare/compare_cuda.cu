#include "hip/hip_runtime.h"
#include <gputk.h>

__global__ void vecAdd(double *in1, double *in2, double out, int len) {
  //@@ Insert code to implement vector addition here
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < len) {
    out += in1[index] + in2[index];
  }
}

int main(int argc, char **argv) {
  gpuTKArg_t args;
  int inputLength;
  double *hostInput1;
  double *hostInput2;
  double hostOutput;
  double *deviceInput1;
  double *deviceInput2;
  double deviceOutput;

  args = gpuTKArg_read(argc, argv);

  gpuTKTime_start(Generic, "Importing data and creating memory on host");
  hostInput1 =
      (double *)gpuTKImport(gpuTKArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (double *)gpuTKImport(gpuTKArg_getInputFile(args, 1), &inputLength);
  gpuTKTime_stop(Generic, "Importing data and creating memory on host");

  gpuTKLog(TRACE, "The input length is ", inputLength);

  gpuTKTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  hipMalloc((void **)&deviceInput1, inputLength * sizeof(double));
  hipMalloc((void **)&deviceInput2, inputLength * sizeof(double));
  gpuTKTime_stop(GPU, "Allocating GPU memory.");

  gpuTKTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(double),
             hipMemcpyHostToDevice);
  gpuTKTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 blockDim(32);
  dim3 gridDim(ceil(((double)inputLength) / ((double)blockDim.x)));

  gpuTKLog(TRACE, "Block dimension is ", blockDim.x);
  gpuTKLog(TRACE, "Grid dimension is ", gridDim.x);

  gpuTKTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  vecAdd<<<gridDim, blockDim>>>(deviceInput1, deviceInput2, deviceOutput,
                                inputLength);
  hipDeviceSynchronize();
  gpuTKTime_stop(Compute, "Performing CUDA computation");

  gpuTKTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, sizeof(double),
             hipMemcpyDeviceToHost);
  gpuTKTime_stop(Copy, "Copying output memory to the CPU");

  gpuTKTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);
  gpuTKTime_stop(GPU, "Freeing GPU Memory");

  gpuTKSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}