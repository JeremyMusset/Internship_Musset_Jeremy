
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <iostream>
#include <float.h>
#include <cmath>
#include <omp.h>
#include <iomanip>
#include <ios>

#define BLOCK_DIM 16
#define NB_EXEC 20

__global__ void seq_dot_prod(double *in1, double *in2, double *out, int size) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  // Stocker le résultat final dans la mémoire globale
  if (index == 0) {
    out[0] = 0;
    for (unsigned int j=0; j<size;j++){
      out[0]+= in1[j]*in2[j];
    }
  }
}

__global__ void par_dot_prod_2(double *in1, double *in2, double *out, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int gridSize = blockDim.x * gridDim.x;
    const int size_local = size;

    double sum_local = 0.0;
    __shared__ double reduction_bloc [1000];

    // Each threads compute local multiplication 
    for (int i = index; i < size; i += gridSize) {
      sum_local += in1[i] * in2[i];
    }

    // Each threads sum his local multiplication to give a local dot product to the thread 0 of his block
    reduction_bloc[threadIdx.x] = sum_local;
    __syncthreads();

    // each block compute the reduction of local results inside his block       
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
      if (threadIdx.x < s) {
        reduction_bloc[threadIdx.x] += reduction_bloc[threadIdx.x + s];
      }
      __syncthreads();
    }
    
    // Each threads 0 save the reduction of his block in out
    if (threadIdx.x == 0) {
      out[blockIdx.x] = reduction_bloc[0];
    }
    __syncthreads();

    // Thread master sum every blocs reductions
    if (blockIdx.x == 0) {
        double final_sum = 0.0;
        for (int i = 0; i < gridDim.x; i++) {
            final_sum += out[i];
        }
        out[0] = final_sum;
    }
}

__global__ void par_dot_prod(double *in1, double *in2, double *out, int size) {
  //@@ Insert code to implement vector addition here
    int index = threadIdx.x;

    // Create shared table
    __shared__ double partials[BLOCK_DIM];
    double partial = 0.0;

    // Each threads compute a part of dot product 
    for (int i = index; i < size; i += blockDim.x) {
        partial += in1[i] * in2[i];
    }
    partials[index] = partial;
    __syncthreads();

    // Reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (index < stride) {
            partials[index] += partials[index + stride];
        }
    __syncthreads();
    }

    // Share result
    if (index == 0) {
        out[0] = partials[0];
    }
}



int main() {
  double *hostInput1;
  double *hostInput2;
  double *hostOutput;
  double *hostOutput2;
  double *deviceInput1;
  double *deviceInput2;
  double *deviceOutput;
  double *deviceOutput2;
  int size;
  int sz_time = 20;
  // double *VSize;
  double coeff = 0;
  double  *Time_cuda_par;
  double  *Time_cuda_seq;
  double  Min_Time_cuda_par;
  double  Min_Time_cuda_seq;
  Time_cuda_par =
      (double *)malloc(20 * sizeof(double));
  Time_cuda_seq =
      (double *)malloc(20 * sizeof(double));
  double *Vec;
  double time_tmp;
  int size_vec;
  int VSize[] = {5000,10000,15000,20000,25000,30000,40000,50000,75000, 100000,200000,300000,400000,500000,625000,750000,875000,1000000,1500000,2000000};
  for(unsigned int k=0; k<sz_time;k++)
  {
  Time_cuda_par[k] = 0;
  Time_cuda_seq[k] = 0;
  size = VSize[k];
  size_vec = 2*size+1;
  for (unsigned int l =0; l<10; l++){
  printf("\n--------------  size : %d --------------- data : %d ---------------- \n\n",size,l);
  // Malloc
  hostInput1 =
      (double *)malloc(size * sizeof(double));
  hostInput2 =
      (double *)malloc(size * sizeof(double));
  hostOutput =
      (double *)malloc( sizeof(double));
  hostOutput2 =
      (double *)malloc( sizeof(double));
  Vec =
      (double *)malloc(size_vec * sizeof(double));


  //@@Import host data
  FILE *fichier;
  char name[500];
  sprintf(name,"../data/vector%d.bin",l);
  fichier = fopen(name, "rb");
  // fichier = fopen("../data/vector0.bin", "rb");
  if (fichier == NULL) {
      printf("Impossible d'ouvrir le fichier.\n");
      exit(EXIT_FAILURE);
  }
  fread(Vec, 8, size_vec, fichier);
  fclose(fichier);
  for (unsigned int i=0;i<size;i++){
        hostInput1[i] = Vec[i+1];
        hostInput2[i] = Vec[size+1+i]; 
    }

  //@@Compute solution 
  struct timespec start_cuda_par, end_cuda_par,start_cuda_seq,end_cuda_seq;
  double result = 0.0;
  for (unsigned int j=0; j<size;j++){
      result += hostInput1[j]*hostInput2[j];
  }
  printf("GOOD RESULT : %.20f \n",result);


  //@@ Allocate GPU memory here
  hipMalloc((void **)&deviceInput1, size * sizeof(double));
  hipMalloc((void **)&deviceInput2, size * sizeof(double));
  hipMalloc((void **)&deviceOutput,sizeof(double));
hipMalloc((void **)&deviceOutput2, sizeof(double));

  //@@ Copy memory to the GPU here
  hipMemcpy(deviceInput1, hostInput1, size * sizeof(double),
             hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, size * sizeof(double),
             hipMemcpyHostToDevice);


  //@@ Initialize the grid and block dimensions here
  dim3 blockDim(BLOCK_DIM);
  dim3 gridDim(ceil((double)1920 / BLOCK_DIM));

  dim3 blockDim2(1);
  dim3 gridDim2(1);
  

  //@@ Launch the GPU Kernel here
  //Parallel
  // gpuTKTime_stop(Compute, "PARALLEL TIME : ");
  par_dot_prod_2<<<gridDim, blockDim>>>(deviceInput1, deviceInput2, deviceOutput,
                                  size);
  // gpuTKTime_stop(Compute, "PARALLEL TIME : ");
  for (unsigned int w =0;w<NB_EXEC;w++){ 
    clock_gettime(CLOCK_REALTIME,&start_cuda_par); 
    par_dot_prod_2<<<gridDim, blockDim>>>(deviceInput1, deviceInput2, deviceOutput,
                                        size);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_REALTIME,&end_cuda_par); 
    time_tmp = ((end_cuda_par.tv_sec - start_cuda_par.tv_sec) * 1000000000 + (end_cuda_par.tv_nsec - start_cuda_par.tv_nsec) );
    if (w==0){
      Min_Time_cuda_par = time_tmp;
    }
      if (time_tmp < Min_Time_cuda_par){
      Min_Time_cuda_par = time_tmp;
    }
  }
  Time_cuda_par[k] += Min_Time_cuda_par;

  hipDeviceSynchronize();

  // Sequential 
  seq_dot_prod<<<gridDim2, blockDim2>>>(deviceInput1, deviceInput2, deviceOutput2,
                                      size);

  for (unsigned int w =0;w<NB_EXEC;w++){ 
    clock_gettime(CLOCK_REALTIME,&start_cuda_seq); 
    seq_dot_prod<<<gridDim2, blockDim2>>>(deviceInput1, deviceInput2, deviceOutput2,
                                        size);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_REALTIME,&end_cuda_seq); 
    time_tmp = ((end_cuda_seq.tv_sec - start_cuda_seq.tv_sec) * 1000000000 + (end_cuda_seq.tv_nsec - start_cuda_seq.tv_nsec) );
    if (w==0){
      Min_Time_cuda_seq = time_tmp;
    }
      if (time_tmp < Min_Time_cuda_seq){
      Min_Time_cuda_seq = time_tmp;
    }
  }  
  Time_cuda_seq[k] += Min_Time_cuda_seq;
  hipDeviceSynchronize();


  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, sizeof(double),
             hipMemcpyDeviceToHost);
  hipMemcpy(hostOutput2, deviceOutput2, sizeof(double),
             hipMemcpyDeviceToHost);

  printf("Cuda par result : %.20f in %.20f\n",hostOutput[0],Min_Time_cuda_par);
  printf("Cuda seq result : %.20f in %.20f\n",hostOutput2[0],Min_Time_cuda_seq);
  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);


  //@@ Free the CPU memory here
  free(hostInput1);
  free(hostInput2);
  }
  Time_cuda_par[k] /= 10;
  Time_cuda_seq[k] /= 10;
  coeff += Time_cuda_seq[k] / Time_cuda_par[k];
  }
  coeff /= sz_time;

  // Prints
  printf("\n\n Average coefficient = %.20f",coeff);
  printf("\n\n Time_cuda_par = {");
  for (unsigned int o = 0;o<19;o++){
    printf(" %.15f, ",Time_cuda_par[o]);
  }
  printf(" %.15f }; \n",Time_cuda_par[19]);

  printf("\n\n Time_cuda_seq = {");
  for (unsigned int o = 0;o<19;o++){
    printf(" %.15f, ",Time_cuda_seq[o]);
  }
  printf(" %.15f }; \n",Time_cuda_seq[19]);
  return 0;
}